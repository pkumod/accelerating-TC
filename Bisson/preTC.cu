#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <vector>
#include <iterator>
#include <fstream>
#include <math.h>
#include <unistd.h>
#include <stdlib.h>
#include <string>
#include <stdio.h>
#include <cmath>
#include<sys/stat.h>
#include<ctime>

#include <hip/hip_runtime.h>

#include<thrust/reduce.h>
#include<hip/hip_runtime.h>
#include<thrust/sort.h>
#include<thrust/device_ptr.h>
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>
#include<thrust/copy.h>
#include<thrust/execution_policy.h>
#include<thrust/scan.h>
using namespace std;
#define thrustSortBlockSize 4000000000
#define bucketNum 10
struct edge{
    unsigned int src;
    unsigned int dst;
};

struct cmpStruc{
    __device__ bool operator () (const edge &a, const edge &b){
        return (a.src < b.src) || (a.src == b.src && a.dst < b.dst) ;
    }
}cmp;
class edgeVector{
		public:
				unsigned int capcity;
				unsigned int esize;
				edge *Edges;
				edgeVector(){esize = 0; capcity = 0;}
				void init(unsigned int s) { Edges = new edge [s]; capcity = s; return ;}
				void addEdge(edge * E){
						if(esize >= capcity) {
								capcity *= 2;
								edge* tmpEdges = new edge [capcity];
								memcpy(tmpEdges,Edges,sizeof(edge)*esize);
								delete [] Edges;
								Edges = tmpEdges;
						}
						memcpy(Edges+esize,E,sizeof(edge));
						esize ++;
				}
				void clear() {delete [] Edges; return ;}
};
unsigned int *edgeOffset;
int *nonZeroRow;
int *edgeRow;
int *adjLength;
edge *Edges;
clock_t start_, end_;
__global__ void degreeCount(int curEdgeNum,edge *degreeCountEdges,int *degreeRecord) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int step = blockDim.x*gridDim.x;
	while (idx < curEdgeNum) {
		int src = degreeCountEdges[idx].src;
		int dst = degreeCountEdges[idx].dst;
		atomicAdd(degreeRecord + src,1);
		atomicAdd(degreeRecord + dst,1);
		idx += step;
	}
	return ;
}
bool preProcess(const char *fileName, unsigned int  &_edgeNum, unsigned &_nodeNum, int &_nonZeroSize, int chhooseIndex)
{
    //get file size
	
    ifstream fin1(fileName,ios::in|ios::binary);
    fin1.seekg(0,ios::end);
    streampos Size = fin1.tellg();
    fin1.close();
    long int size = Size;
    cout << "the size of input file is " << size << " Byte. " << endl;
    unsigned int edgeNum = size/(sizeof(int)*2);
    Edges = new edge [edgeNum];

    //read data
    ifstream fin(fileName, std::ios::binary);
    if (fin.bad()) {
		cout << "File not fould!" << endl;
		return false;
	}
    cout << "start read data... ..." << endl;
    fin.read((char *)Edges,sizeof(edge)*edgeNum);
    fin.close();
    cout << "end read data" << endl;

    //pre work
    //fine node number
	unsigned int maxNodeID = 0;
	#pragma omp parallel for reduction(max : maxNodeID)
	for (int i = 0; i < edgeNum; i ++) {
		maxNodeID = max(maxNodeID,Edges[i].src);
		maxNodeID = max(maxNodeID,Edges[i].dst);
	}
	unsigned nodeNum = maxNodeID + 1;
	cout << "end findMax node" << endl;


	//cal degrees
	int * degreeRecord = new int[nodeNum];
	int * d_degreeRecord;
	hipMalloc(&d_degreeRecord,sizeof(int)*nodeNum);
	hipMemset(d_degreeRecord,0,sizeof(int)*nodeNum);
	edge * d_degreeCountEdges;
	int degreeCountBlockSize = 500000000;
	hipMalloc(&d_degreeCountEdges,sizeof(edge)*degreeCountBlockSize);
	for (int i = 0; i < (edgeNum + degreeCountBlockSize -1 )/degreeCountBlockSize; i++) {
		int curEdgeNum = edgeNum - degreeCountBlockSize*i;
		curEdgeNum  = (curEdgeNum > degreeCountBlockSize) ? degreeCountBlockSize: curEdgeNum;
		hipMemcpy((void *)d_degreeCountEdges,Edges+i*degreeCountBlockSize,curEdgeNum*sizeof(edge),hipMemcpyHostToDevice);
		int curBlockSize = (curEdgeNum + 1024 - 1)/1024;
		curBlockSize = (curBlockSize > 60000)? 60000: curBlockSize;
		degreeCount<<<curBlockSize,1024>>>(curEdgeNum,d_degreeCountEdges,d_degreeRecord);
	}
	hipMemcpy((void *)degreeRecord,(void *)d_degreeRecord,sizeof(int)*nodeNum,hipMemcpyDeviceToHost);
	hipFree(d_degreeRecord);
	hipFree(d_degreeCountEdges);

//	#pragma omp parallel for
	double start_t = omp_get_wtime();
	if (chhooseIndex == 0) {
		for (unsigned int i = 0; i < edgeNum; i ++) {
			unsigned int src = Edges[i].src;
			unsigned int dst = Edges[i].dst;
			if (src < dst) {
		//if (degreeRecord[src] > degreeRecord[dst] || (degreeRecord[src] == degreeRecord[dst] && src < dst)) {
				Edges[i].src = dst;
				Edges[i].dst = src;
			}
		}
	} else {
		unsigned int * srcp = new unsigned [nodeNum];
		unsigned int * dstp = new unsigned [nodeNum];
		for (int i = 0; i < nodeNum; i ++)
			srcp[i] = i;
		thrust::sort_by_key(degreeRecord,degreeRecord+nodeNum,srcp);
		for (int i = 0; i< nodeNum; i ++)
			dstp[srcp[i]] = nodeNum-1-i;
//#pragma omp parallel for
		for (int i = 0; i < edgeNum; i ++) {
			int src = dstp[Edges[i].src];
			int dst = dstp[Edges[i].dst];
			Edges[i].src = max(src,dst);
			Edges[i].dst = min(src,dst);
		}
		delete [] srcp;
		delete [] dstp;
	}
	double end_t = omp_get_wtime();
	std::cout << "the total time of edge direction is " << (end_t - start_t) << " s. " << endl;
	delete [] degreeRecord;	
    //sort edges
	//************sort edges********
 	edgeVector * edgeBucket = new edgeVector [bucketNum];
	for (int i = 0; i < bucketNum; i ++) 
		edgeBucket[i].init(edgeNum/bucketNum);
	unsigned bucketStep = (nodeNum + bucketNum - 1)/bucketNum; 
	for (int i = 0; i < edgeNum; i ++)
	{
		int bucketID = Edges[i].src/bucketStep;
		edgeBucket[bucketID].addEdge(Edges+i);
	}
	cout << "end pust edges in bucket" << endl;
	unsigned int *bucketEdgeOffset = new unsigned int [bucketNum];
	bucketEdgeOffset[0] = 0;
	for (int i = 0; i < bucketNum-1; i ++) {
		unsigned int bucketSize = edgeBucket[i].esize;
		if (bucketSize > thrustSortBlockSize/sizeof(edge)) {
			cout << "bucket " << i << "size is " << bucketSize << ", it's too large!" << endl;
			return false;
		}
		bucketEdgeOffset[i+1] = bucketEdgeOffset[i] + bucketSize;
	}
	for (int i = 0; i < bucketNum; i++) {
		thrust::device_vector<edge> D (edgeBucket[i].Edges, edgeBucket[i].Edges+edgeBucket[i].esize);
		thrust::sort(D.begin(),D.begin()+edgeBucket[i].esize,cmp);
		thrust::copy(D.begin(),D.begin()+edgeBucket[i].esize,edgeBucket[i].Edges);
	}
	cout << "end sort edges in GPU " << endl;
	for(int i = 0; i < bucketNum; i ++) {
		memcpy(Edges+bucketEdgeOffset[i],edgeBucket[i].Edges,sizeof(edge)*edgeBucket[i].esize);
	}
	cout << "end copy result to Edges" << endl;
	delete [] bucketEdgeOffset;
	for (int i = 0; i < bucketNum; i ++)
		edgeBucket[i].clear();
	delete [] edgeBucket;
	//************end sort edges && get nodeNum********

    //unsigned int nodeNum = Edges[edgeNum-1].src + 1;
    edgeOffset = new unsigned int [nodeNum+2];
    edgeOffset[0] = 0;
    edgeRow = new int [edgeNum+1];
    adjLength = new int[nodeNum+1];
	memset(adjLength,0,sizeof(int)*(nodeNum+1));
	unsigned int nodePos = 0;
	unsigned int edgePos = 0;
	edge * edgePtr;
	int formerSrc = -1,formerDst = -1;
	start_ = clock();
//	for (int i = 0; i < edgeNum; i++)
//		printf("%d   %d\n",Edges[i].src,Edges[i].dst);
	for (unsigned int i = 0; i < edgeNum; i++)
	{
			edgePtr = Edges + i;
			if (edgePtr->src == edgePtr->dst) {
					formerSrc = edgePtr->src;
					formerDst = edgePtr->dst;
					int curSrc = edgePtr->src;
					for (unsigned j = nodePos + 1; j <= curSrc; j++) {
							edgeOffset[j] = edgePos;
							adjLength[j-1] = edgeOffset[j]-edgeOffset[j-1];	
					}
					nodePos = curSrc;
					continue;
			}
			if ((i > 0) && (edgePtr->src == formerSrc)) {
					//TODO  find a more efficienty way
					if(edgePtr->dst == formerDst){
							continue;
					}
					edgeRow[edgePos++] = edgePtr->dst;
					formerDst = edgePtr->dst;
					continue;
			}	
			int curSrc = edgePtr->src;
			for (unsigned j = nodePos + 1; j <= curSrc; j++) {
					edgeOffset[j] = edgePos;
					adjLength[j-1] = edgeOffset[j]-edgeOffset[j-1];	
			}
			nodePos = curSrc;
			edgeRow[edgePos++] = edgePtr->dst;
			formerSrc = edgePtr->src;
			formerDst = edgePtr->dst;
//	    cout << " end an edge in a loop " << endl;
    }
	for (unsigned i = nodePos + 1; i < nodeNum; i ++) {
			edgeOffset[i] = edgePos;
			adjLength[i-1] = edgeOffset[i] - edgeOffset[i-1];
	}
	end_ = clock();
	cout << "merge and make csr use " << (double)1000*(end_-start_)/CLOCKS_PER_SEC << " ms." << endl;
    edgeOffset[nodeNum] = edgePos;
	edgeOffset[nodeNum+1] = edgePos + 1;
    adjLength[nodeNum-1] = edgeOffset[nodeNum] - edgeOffset[nodeNum-1];
	adjLength[nodeNum] = 1024;
	edgeRow[edgePos] = nodeNum;
    cout << "csr built, edgeNum is "<< edgePos<< ", the node num is " << nodeNum << ", origin egde num is " << edgeNum << endl;
    //TODO remove empty node in edgeOffset
    _edgeNum = edgeOffset[nodeNum];
    _nodeNum = nodeNum;
	
	unsigned int nonZeroSize = 0;
	nonZeroRow = new int [nodeNum];
	for (int i = 0; i < nodeNum; i ++) {
		if (edgeOffset[i] != edgeOffset[i+1]) {
			nonZeroRow[nonZeroSize++] = i;
		//	cout << "degree " << i << " is " << edgeOffset[i+1]-edgeOffset[i] << endl;
		}
	}
	int *tmpNonZeroRow = new int [nonZeroSize];
	memcpy(tmpNonZeroRow,nonZeroRow,sizeof(int)*nonZeroSize);
	delete [] nonZeroRow;
	nonZeroRow = tmpNonZeroRow;
	_nonZeroSize = nonZeroSize;
	cout << "the nonZeroSize is " << nonZeroSize << endl;
/*	for (int i = 0; i <= _nodeNum+1;  i++)
		cout << edgeOffset[i] << " ";
	cout << endl;
	for (int i = 0; i < _nodeNum;  i++)
		cout << adjLength[i] << " ";
	cout << endl;
	for (int i = 0; i < _edgeNum; i ++)
		cout << edgeRow[i] << " ";
	cout << endl;
*/
    delete [] Edges;
    return true;
}
